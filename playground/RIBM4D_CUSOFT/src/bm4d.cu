#include "hip/hip_runtime.h"
/*
 * 2016, Vladislav Tananaev
 * v.d.tananaev [at] gmail [dot] com
 * 2020, Xingyu Zhu
 * jupiter.zhuxingyu [at] gmail [dot] com
 */

#include <bm4d-gpu/bm4d.h>
// texture<uchar, 3, hipReadModeNormalizedFloat> noisy_volume_3d_tex;

void BM4D::load_3d_array() {
    Stopwatch copyingtodevice(true);
    const hipExtent volumeSize = make_hipExtent(width, height, depth);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    checkCudaErrors(hipMalloc3DArray(&d_noisy_volume_3d, &channelDesc, volumeSize));

    // Copy data to 3D array (host to device)
    uchar *volume_tmp = &noisy_volume[0];
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void*)volume_tmp, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_noisy_volume_3d;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));
    copyingtodevice.stop(); std::cout << "Copying to device (3d tex) took:" << copyingtodevice.getSeconds() << std::endl;
    
    // Create the surface object
    struct hipResourceDesc surfRes;
    memset(&surfRes, 0, sizeof(surfRes));
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = d_noisy_volume_3d;
    checkCudaErrors(hipCreateSurfaceObject(&noisy_volume_3d_surf, &surfRes));
    std::cout << "Binded with surface reference" << std::endl;
}

void BM4D::init_masks() {
    float std = pshift * 0.75; // std of gaussian 
    float sphere_tol = (pshift + 0.25) * (pshift + 0.25); // max distance to be included in the sphere
    int k = params.patch_size;

    Stopwatch t_init_mask(true);
    maskGaussian = (float*) malloc(psize * sizeof(float));
    maskSphere = (float*) malloc(psize * sizeof(float));
    // Odd size
    float dx, dy, dz, sqr_dist;
    int d;
    for (int z = 0; z < k; ++z)
        for (int y = 0; y < k; ++y)
            for (int x = 0; x < k; ++x) {
                d = x + y * k + z * k * k;
                dx = float(x) - pshift;
                dy = float(y) - pshift;
                dz = float(z) - pshift;
                sqr_dist = dx*dx + dy*dy + dz*dz;
                
                // Gaussian
                maskGaussian[d] = normal_pdf_sqr(std, sqr_dist);
                
                // Sphere
                if (sqr_dist <= sphere_tol) maskSphere[d] = 1.0;
                else maskSphere[d] = 0.0;
            }
    
    checkCudaErrors(hipMalloc((void**)&d_maskGaussian, sizeof(float) * psize));
    checkCudaErrors(hipMemcpy((void*)d_maskGaussian, (void*)maskGaussian, sizeof(float) * psize, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&d_maskSphere, sizeof(float) * psize));
    checkCudaErrors(hipMemcpy((void*)d_maskSphere, (void*)maskSphere, sizeof(float) * psize, hipMemcpyHostToDevice));
    
    t_init_mask.stop(); std::cout<<"Initialize masks took: " << t_init_mask.getSeconds() <<std::endl;
};

std::vector<uchar> BM4D::run_first_step() {
    assert(size == noisy_volume.size());
    checkCudaErrors(hipMemcpy((void*)d_noisy_volume, (void*)noisy_volume.data(), sizeof(uchar) * size, hipMemcpyHostToDevice));
    load_3d_array();
    uint3 imshape = make_uint3(width, height, depth);
    uint3 tshape = make_uint3(twidth, theight, tdepth);    // Truncated size, with some step for ref patches
    d_volume2stack(d_noisy_volume, d_noisy_stacks, imshape, tshape, params, d_prop);
    
    // Pre-compute spehrical representation
    Stopwatch t_pre_comp_fft(true);
    std::cout << "\nComputing spherical representation of patches" << std::endl;
    run_fft_precomp(d_noisy_stacks, imshape, tshape, params, d_sigR, d_sigI, d_prop);
    t_pre_comp_fft.stop();
    std::cout << "took: " << t_pre_comp_fft.getSeconds() << std::endl;
    
    sample_run(d_sigR, d_sigI,
        d_so3SigR, d_so3SigI,
        d_workspace1, d_workspace2,
        d_sigCoefR, d_sigCoefI,
        d_patCoefR, d_patCoefI,
        d_so3CoefR, d_so3CoefI,
        d_seminaive_naive_tablespace,
        d_cos_even,
        d_seminaive_naive_table,
        bwIn, bwOut, degLim,
        sig_patch_size,
        wsp1_bsize,
        wsp2_bsize,
        sigpatCoef_bsize,
        so3Coef_bsize,
        so3Sig_bsize,
        SNTspace_bsize,
        SNT_bsize,
        cos_even_bsize);
    
    
    return noisy_volume;
    // Do block matching
    Stopwatch blockmatching(true);
    std::cout << "\nStart blockmatching" << std::endl;
    run_block_matching(d_noisy_volume, imshape, tshape, params, d_stacks, d_nstacks, d_prop);
    run_block_matching_rot(d_noisy_volume, d_sigR, d_sigI, imshape, tshape, params, d_stacks_rot, d_nstacks_rot, sig_patch_size, d_prop);
    blockmatching.stop();
    std::cout << "Blockmatching took: " << blockmatching.getSeconds() << std::endl;

    // Gather cubes together
    uint gather_stacks_sum;
    Stopwatch gatheringcubes(true);
    gather_cubes(d_noisy_volume, imshape, tshape, params, d_stacks, d_nstacks, d_gathered4dstack, gather_stacks_sum, d_prop);
    // std::cout << "Acquied size " << gather_stacks_sum << std::endl;
    gatheringcubes.stop();
    std::cout << "Gathering cubes took: " << gatheringcubes.getSeconds() << std::endl;
    // debug_kernel(d_gathered4dstack);
    checkCudaErrors(hipFree(d_noisy_volume));

    // Perform 3D DCT
    Stopwatch dct_forward(true);
    run_dct3d(d_gathered4dstack, gather_stacks_sum, params.patch_size, d_prop);
    dct_forward.stop();
    std::cout << "3D DCT forwards took: " << dct_forward.getSeconds() << std::endl;
    // debug_kernel(d_gathered4dstack);

    // Do WHT in 4th dim + Hard Thresholding + IWHT
    float* d_group_weights;
    Stopwatch wht_t(true);
    run_wht_ht_iwht(d_gathered4dstack, gather_stacks_sum, params.patch_size, d_nstacks, tshape, d_group_weights, params, d_prop);
    wht_t.stop();
    std::cout << "WHT took: " << wht_t.getSeconds() << std::endl;

    // Perform inverse 3D DCT
    Stopwatch dct_backward(true);
    run_idct3d(d_gathered4dstack, gather_stacks_sum, params.patch_size, d_prop);
    dct_backward.stop();
    std::cout << "3D DCT backwards took: " << dct_backward.getSeconds() << std::endl;
    // debug_kernel(d_gathered4dstack);


    // Aggregate
    float* final_image = new float[width * height * depth];
    memset(final_image, 0.0, sizeof(float) * width * height * depth);
    Stopwatch aggregation_t(true);
    run_aggregation(final_image, imshape, tshape, d_gathered4dstack, d_stacks, d_nstacks, d_group_weights, params, gather_stacks_sum, d_prop);
    aggregation_t.stop();
    std::cout << "Aggregation took: " << aggregation_t.getSeconds() << std::endl;
    for (int i = 0; i < size; i++) {
        noisy_volume[i] = static_cast<uchar>(final_image[i]);
    }
    delete[] final_image;
    return noisy_volume;
}
