/***************************************************************************
  **************************************************************************
  
  S2kit 1.0
  A lite version of Spherical Harmonic Transform Kit

  Copyright (c) 2004 Peter Kostelec, Dan Rockmore

  This file is part of S2kit.

  S2kit is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 3 of the License, or
  (at your option) any later version.

  S2kit is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
  
  See the accompanying LICENSE file for details.
  
  ************************************************************************
  ************************************************************************/


#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>  /* to declare memcpy */
#include <stdlib.h>

#ifndef PI
#define PI 3.14159265358979
#endif

#define compmult(a,b,c,d,e,f) (e) = ((a)*(c))-((b)*(d)); (f) = ((a)*(d))+((b)*(c))


/************************************************************************/
/* Recurrence coefficients */
/************************************************************************/
/* Recurrence coefficents for L2-normed associated Legendre
   recurrence.  When using these coeffs, make sure that
   inital Pmm function is also L2-normed */
/* l represents degree, m is the order */

__device__ double L2_an(int m,
	     int l)
{
  return (sqrt((((double) (2*l+3))/((double) (2*l+1))) *
	       (((double) (l-m+1))/((double) (l+m+1)))) *
	  (((double) (2*l+1))/((double) (l-m+1))));

}

/* note - if input l is zero, need to return 0 */
__device__ double L2_cn(int m,
	     int l) 
{
  if (l != 0) {
    return (-1.0 *
	  sqrt((((double) (2*l+3))/((double) (2*l-1))) *
	       (((double) (l-m+1))/((double) (l+m+1))) *
	       (((double) (l-m))/((double) (l+m)))) *
	  (((double) (l+m))/((double) (l-m+1))));
  }
  else
    return 0.0;

}

/************************************************************************/
/* vector arithmetic operations */
/************************************************************************/
/* does result = data1 + data2 */
/* result and data are vectors of length n */

__device__ void vec_add(double *data1,
	     double *data2,
	     double *result,
	     int n)
{
  int k;

  for (k = 0; k < n % 4; ++k)
    result[k] = data1[k] + data2[k];

  for ( ; k < n ; k += 4)
    {
      result[k] = data1[k] + data2[k];
      result[k + 1] = data1[k + 1] + data2[k + 1];
      result[k + 2] = data1[k + 2] + data2[k + 2];
      result[k + 3] = data1[k + 3] + data2[k + 3];
    }
}
/************************************************************************/
/************************************************************************/
/*
   vec_mul(scalar,data1,result,n) multiplies the vector 'data1' by
   'scalar' and returns in result 
*/
__device__ void vec_mul(double scalar,
	     double *data1,
	     double *result,
	     int n)
{
   int k;


   for( k = 0; k < n % 4; ++k)
     result[k] = scalar * data1[k];

   for( ; k < n; k +=4)
     {
       result[k] = scalar * data1[k];
       result[k + 1] = scalar * data1[k + 1];
       result[k + 2] = scalar * data1[k + 2];
       result[k + 3] = scalar * data1[k + 3];
     }

}
/************************************************************************/
/* point-by-point multiplication of vectors */

__device__ void vec_pt_mul(double *data1,
		double *data2,
		double *result,
		int n)
{
   int k;
  
  for(k = 0; k < n % 4; ++k)
    result[k] = data1[k] * data2[k];
  
  for( ; k < n; k +=4)
    {
      result[k] = data1[k] * data2[k];
      result[k + 1] = data1[k + 1] * data2[k + 1];
      result[k + 2] = data1[k + 2] * data2[k + 2];
      result[k + 3] = data1[k + 3] * data2[k + 3];
    }
 
}


/************************************************************************/
/* returns an array of the angular arguments of n Chebyshev nodes */
/* eval_pts points to a double array of length n */

__device__ void ArcCosEvalPts(int n,
		   double *eval_pts)
{
    int i;
    double twoN;

    twoN = (double) (2 * n);

   for (i=0; i<n; i++)
     eval_pts[i] = (( 2.0*((double)i)+1.0 ) * PI) / twoN;

}
/************************************************************************/
/* returns an array of n Chebyshev nodes */

__device__ void EvalPts( int n,
	      double *eval_pts)
{
    int i;
    double twoN;

    twoN = (double) (2*n);

   for (i=0; i<n; i++)
     eval_pts[i] = cos((( 2.0*((double)i)+1.0 ) * PI) / twoN);

}

/************************************************************************/
/* L2 normed Pmm.  Expects input to be the order m, an array of
 evaluation points arguments of length n, and a result vector of length n */
/* The norming constant can be found in Sean's PhD thesis */
/* This has been tested and stably computes Pmm functions thru bw=512 */

__device__ void Pmm_L2( int m,
	     double *eval_pts,
	     int n,
	     double *result)
{
  int i;
  double md, id, mcons;

  id = (double) 0.0;
  md = (double) m;
  mcons = sqrt(md + 0.5);

  for (i=0; i<m; i++) {
    mcons *= sqrt((md-(id/2.0))/(md-id));
    id += 1.0;
  }
  if (m != 0 )
    mcons *= pow(2.0,-md/2.0);
  if ((m % 2) != 0) mcons *= -1.0;

  for (i=0; i<n; i++) 
    result[i] = mcons * pow(sin(eval_pts[i]),md);

}

