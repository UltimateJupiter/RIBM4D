#include "hip/hip_runtime.h"
// Simple copy kernel
__global__ void copyKernel(hipSurfaceObject_t inputSurfObj,
                           hipSurfaceObject_t outputSurfObj,
                           int width, int height) 
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uchar4 data;
        // Read from input surface
        surf2Dread(&data,  inputSurfObj, x * 4, y);
        // Write to output surface
        surf2Dwrite(data, outputSurfObj, x * 4, y);
    }
}

// Host code
int main()
{
    // Allocate CUDA arrays in device memory
    hipChannelFormatDesc channelDesc =
             hipCreateChannelDesc(8, 8, 8, 8,
                                   hipChannelFormatKindUnsigned);
    hipArray* cuInputArray;
    hipMallocArray(&cuInputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);
    hipArray* cuOutputArray;
    hipMallocArray(&cuOutputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);

    // Copy to device memory some data located at address h_data
    // in host memory 
    hipMemcpyToArray(cuInputArray, 0, 0, h_data, size,
                      hipMemcpyHostToDevice);

    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    // Create the surface objects
    resDesc.res.array.array = cuInputArray;
    hipSurfaceObject_t inputSurfObj = 0;
    hipCreateSurfaceObject(&inputSurfObj, &resDesc);
    resDesc.res.array.array = cuOutputArray;
    hipSurfaceObject_t outputSurfObj = 0;
    hipCreateSurfaceObject(&outputSurfObj, &resDesc);

    // Invoke kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y);
    copyKernel<<<dimGrid, dimBlock>>>(inputSurfObj,
                                      outputSurfObj,
                                      width, height);


    // Destroy surface objects
    hipDestroySurfaceObject(inputSurfObj);
    hipDestroySurfaceObject(outputSurfObj);

    // Free device memory
    hipFreeArray(cuInputArray);
    hipFreeArray(cuOutputArray);

    return 0;
}

#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#define NUM_TEX 4

const int SizeNoiseTest = 32;
const int cubeSizeNoiseTest = SizeNoiseTest*SizeNoiseTest*SizeNoiseTest;
static hipTextureObject_t texNoise[NUM_TEX];

__global__ void AccesTexture(hipTextureObject_t my_tex)
{
    float test = tex3D<float>(my_tex,(float)threadIdx.x,(float)threadIdx.y,(float)threadIdx.z);//by using this the error occurs
    printf("thread: %d,%d,%d, value: %f\n", threadIdx.x, threadIdx.y, threadIdx.z, test);
}

void CreateTexture()
{

    float *d_NoiseTest;//Device Array with random floats
    hipMalloc((void **)&d_NoiseTest, cubeSizeNoiseTest*sizeof(float));//Allocation of device Array
    for (int i = 0; i < NUM_TEX; i++){
    //hiprand Random Generator (needs compiler link -lcurand)
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,1235ULL+i);
    hiprandGenerateUniform(gen, d_NoiseTest, cubeSizeNoiseTest);//writing data to d_NoiseTest
    hiprandDestroyGenerator(gen);

    //hipArray Descriptor
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    //cuda Array
    hipArray *d_cuArr;
    checkCudaErrors(hipMalloc3DArray(&d_cuArr, &channelDesc, make_hipExtent(SizeNoiseTest*sizeof(float),SizeNoiseTest,SizeNoiseTest), 0));
    hipMemcpy3DParms copyParams = {0};


    //Array creation
    copyParams.srcPtr   = make_hipPitchedPtr(d_NoiseTest, SizeNoiseTest*sizeof(float), SizeNoiseTest, SizeNoiseTest);
    copyParams.dstArray = d_cuArr;
    copyParams.extent   = make_hipExtent(SizeNoiseTest,SizeNoiseTest,SizeNoiseTest);
    copyParams.kind     = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));
    //Array creation End

    hipResourceDesc    texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array  = d_cuArr;
    hipTextureDesc     texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;   // clamp
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;
    checkCudaErrors(hipCreateTextureObject(&texNoise[i], &texRes, &texDescr, NULL));}
}

int main(int argc, char **argv)
{
    CreateTexture();
    AccesTexture<<<1,dim3(2,2,2)>>>(texNoise[0]);
    AccesTexture<<<1,dim3(2,2,2)>>>(texNoise[1]);
    AccesTexture<<<1,dim3(2,2,2)>>>(texNoise[2]);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    return 0;
}