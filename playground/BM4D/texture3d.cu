#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <fstream> 

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

typedef unsigned char uchar;

#define BLOCKSIZE 16

float w = 0.5;  // texture coordinate in z

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code); }
    }
}

typedef unsigned int  uint;
typedef unsigned char uchar;

texture<uchar, 3, hipReadModeNormalizedFloat> tex;  // 3D texture

hipArray *d_volumeArray = 0;

uint *d_output = NULL;
uint *h_output = NULL;

/************************************************/
/* TEXTURE-BASED TRILINEAR INTERPOLATION KERNEL */
/************************************************/
__global__ void
d_render(uint *d_output, uint imageW, uint imageH, float w)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    float u = x / (float) imageW;
    float v = y / (float) imageH;

    // read from 3D texture
    float voxel = tex3D(tex, u, v, w);

    if ((x < imageW) && (y < imageH)) {
        // write output color
        uint i = __umul24(y, imageW) + x;
        d_output[i] = voxel*255;
   }
}

void main() {

    int N = 32;
    int imageH = 512;
    int imageW = 512;

    const char* filename = "Bucky.raw";

    // --- Loading data from file
    FILE *fp = fopen(filename, "rb");
    if (!fp) { fprintf(stderr, "Error opening file '%s'\n", filename); getchar(); return; }

    uchar *data = (uchar*)malloc(N*N*N*sizeof(uchar));
    size_t read = fread(data, 1, N*N*N, fp);
    fclose(fp);

    printf("Read '%s', %lu bytes\n", filename, read);

    gpuErrchk(hipMalloc((void**)&d_output, imageH*imageW*sizeof(uint)));

    // --- Create 3D array
    const hipExtent volumeSize = make_hipExtent(N, N, N);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    gpuErrchk(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // --- Copy data to 3D array (host to device)
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void*)data, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    gpuErrchk(hipMemcpy3D(&copyParams));

    // --- Set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    tex.addressMode[1] = hipAddressModeWrap;
    tex.addressMode[2] = hipAddressModeWrap;

    // --- Bind array to 3D texture
    gpuErrchk(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

    // --- Launch the interpolation kernel
    const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
    const dim3 gridSize(imageW / blockSize.x, imageH / blockSize.y);
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, w);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // --- Copy the interpolated data to host
    h_output = (uint*)malloc(imageW*imageH*sizeof(uint));
    gpuErrchk(hipMemcpy(h_output,d_output,imageW*imageH*sizeof(uint),hipMemcpyDeviceToHost));

    std::ofstream outfile;
    outfile.open("out_texture.dat", std::ios::out | std::ios::binary);
    outfile.write((char*)h_output, imageW*imageH*sizeof(uint));
    outfile.close();

    getchar();

}