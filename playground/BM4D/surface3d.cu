#include "hip/hip_runtime.h"
// Simple copy kernel
__global__ void copyKernel(hipSurfaceObject_t inputSurfObj,
                           hipSurfaceObject_t outputSurfObj,
                           int width, int height) 
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uchar4 data;
        // Read from input surface
        surf2Dread(&data,  inputSurfObj, x * 4, y);
        // Write to output surface
        surf2Dwrite(data, outputSurfObj, x * 4, y);
    }
}

// Host code
int main()
{
    // Allocate CUDA arrays in device memory
    hipChannelFormatDesc channelDesc =
             hipCreateChannelDesc(8, 8, 8, 8,
                                   hipChannelFormatKindUnsigned);
    hipArray* cuInputArray;
    hipMallocArray(&cuInputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);
    hipArray* cuOutputArray;
    hipMallocArray(&cuOutputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);

    // Copy to device memory some data located at address h_data
    // in host memory 
    hipMemcpyToArray(cuInputArray, 0, 0, h_data, size,
                      hipMemcpyHostToDevice);

    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    // Create the surface objects
    resDesc.res.array.array = cuInputArray;
    hipSurfaceObject_t inputSurfObj = 0;
    hipCreateSurfaceObject(&inputSurfObj, &resDesc);
    resDesc.res.array.array = cuOutputArray;
    hipSurfaceObject_t outputSurfObj = 0;
    hipCreateSurfaceObject(&outputSurfObj, &resDesc);

    // Invoke kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y);
    copyKernel<<<dimGrid, dimBlock>>>(inputSurfObj,
                                      outputSurfObj,
                                      width, height);


    // Destroy surface objects
    hipDestroySurfaceObject(inputSurfObj);
    hipDestroySurfaceObject(outputSurfObj);

    // Free device memory
    hipFreeArray(cuInputArray);
    hipFreeArray(cuOutputArray);

    return 0;
}