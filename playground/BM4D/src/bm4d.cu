#include "hip/hip_runtime.h"
/*
 * 2016, Vladislav Tananaev
 * v.d.tananaev [at] gmail [dot] com
 * 2020, Xingyu Zhu
 * jupiter.zhuxingyu [at] gmail [dot] com
 */

#include <bm4d-gpu/bm4d.h>
// texture<uchar, 3, hipReadModeNormalizedFloat> noisy_volume_3d_tex;

void BM4D::load_3d_array() {
    Stopwatch copyingtodevice(true);
    const hipExtent volumeSize = make_hipExtent(width, height, depth);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    checkCudaErrors(hipMalloc3DArray(&d_noisy_volume_3d, &channelDesc, volumeSize));

    // Copy data to 3D array (host to device)
    uchar *volume_tmp = &noisy_volume[0];
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void*)volume_tmp, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_noisy_volume_3d;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));
    copyingtodevice.stop(); std::cout << "Copying to device (3d tex) took:" << copyingtodevice.getSeconds() << std::endl;
    
    // Create the surface object
    struct hipResourceDesc surfRes;
    memset(&surfRes, 0, sizeof(surfRes));
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = d_noisy_volume_3d;
    checkCudaErrors(hipCreateSurfaceObject(&noisy_volume_3d_surf, &surfRes));
    std::cout << "Binded with surface reference" << std::endl;
}

std::vector<uchar> BM4D::run_first_step() {
    uchar* d_noisy_volume;
    assert(size == noisy_volume.size());
    load_3d_array();
    checkCudaErrors(hipMalloc((void**)&d_noisy_volume, sizeof(uchar) * size));
    checkCudaErrors(hipMemcpy((void*)d_noisy_volume, (void*)noisy_volume.data(), sizeof(uchar) * size, hipMemcpyHostToDevice));

    uint3 imshape = make_uint3(width, height, depth);
    uint3 tshape = make_uint3(twidth, theight, tdepth);    // Truncated size, with some step for ref patches
    
    // Pre-compute spehrical representation
    Stopwatch t_pre_comp_fft(true);
    std::cout << "\nComputing spherical representation of patches" << std::endl;
    run_fft_precomp(d_noisy_volume, imshape, tshape, params, d_shfft_res, d_prop);
    t_pre_comp_fft.stop();
    std::cout << "took: " << t_pre_comp_fft.getSeconds() << std::endl;

    // Do block matching
    Stopwatch blockmatching(true);
    std::cout << "\nStart blockmatching" << std::endl;
    run_block_matching(d_noisy_volume, imshape, tshape, params, d_stacks, d_nstacks, d_prop);
    run_block_matching_rot(d_noisy_volume, d_shfft_res, imshape, tshape, params, d_stacks_rot, d_nstacks_rot, fft_patch_size, d_prop);
    blockmatching.stop();
    std::cout << "Blockmatching took: " << blockmatching.getSeconds() << std::endl;

    // Gather cubes together
    uint gather_stacks_sum;
    Stopwatch gatheringcubes(true);
    gather_cubes(d_noisy_volume, imshape, tshape, params, d_stacks, d_nstacks, d_gathered4dstack, gather_stacks_sum, d_prop);
    // std::cout << "Acquied size " << gather_stacks_sum << std::endl;
    gatheringcubes.stop();
    std::cout << "Gathering cubes took: " << gatheringcubes.getSeconds() << std::endl;
    // debug_kernel(d_gathered4dstack);
    checkCudaErrors(hipFree(d_noisy_volume));

    // Perform 3D DCT
    Stopwatch dct_forward(true);
    run_dct3d(d_gathered4dstack, gather_stacks_sum, params.patch_size, d_prop);
    dct_forward.stop();
    std::cout << "3D DCT forwards took: " << dct_forward.getSeconds() << std::endl;
    // debug_kernel(d_gathered4dstack);

    // Do WHT in 4th dim + Hard Thresholding + IWHT
    float* d_group_weights;
    Stopwatch wht_t(true);
    run_wht_ht_iwht(d_gathered4dstack, gather_stacks_sum, params.patch_size, d_nstacks, tshape, d_group_weights, params, d_prop);
    wht_t.stop();
    std::cout << "WHT took: " << wht_t.getSeconds() << std::endl;

    // Perform inverse 3D DCT
    Stopwatch dct_backward(true);
    run_idct3d(d_gathered4dstack, gather_stacks_sum, params.patch_size, d_prop);
    dct_backward.stop();
    std::cout << "3D DCT backwards took: " << dct_backward.getSeconds() << std::endl;
    // debug_kernel(d_gathered4dstack);
    return noisy_volume;

    // Aggregate
    float* final_image = new float[width * height * depth];
    memset(final_image, 0.0, sizeof(float) * width * height * depth);
    Stopwatch aggregation_t(true);
    run_aggregation(final_image, imshape, tshape, d_gathered4dstack, d_stacks, d_nstacks, d_group_weights, params, gather_stacks_sum, d_prop);
    aggregation_t.stop();
    std::cout << "Aggregation took: " << aggregation_t.getSeconds() << std::endl;
    for (int i = 0; i < size; i++) {
        noisy_volume[i] = static_cast<uchar>(final_image[i]);
    }
    delete[] final_image;
    return noisy_volume;
}
