/***************************************************************************
  **************************************************************************
    
  Spherical Harmonic Transform Kit 2.7
    
  Copyright 1997-2003  Sean Moore, Dennis Healy,
                       Dan Rockmore, Peter Kostelec
  Copyright 2004  Peter Kostelec, Dan Rockmore

  This file is part of SpharmonicKit.

  SpharmonicKit is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 3 of the License, or
  (at your option) any later version.

  SpharmonicKit is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
  
  See the accompanying LICENSE file for details.

  ************************************************************************
  ************************************************************************/

/* indextables.c - source code to hard code bit reverse permutations */


#include <hip/hip_runtime.h>
#include <string.h> /* for memcpy */

__device__ __constant__ int p2[2] = {0,1};

__device__ __constant__ int p4[4] = {0,2,1,3};

__device__ __constant__ int p8[8] = {0, 4, 2, 6, 1, 5, 3, 7};

__device__ __constant__ int p16[16] = {0, 8, 4, 12, 2, 10, 6, 14, 1, 9, 5, 13, 3, 11, 7, 15};

__device__ __constant__ int  p32[32] =
{0, 16, 8, 24, 4, 20, 12, 28, 2, 18, 10, 26, 6, 22, 14, 30, 1, 
 17, 9, 25, 5, 21, 13, 29, 3, 19, 11, 27, 7, 23, 15, 31};

__device__ __constant__ int  p64[64] =
{0, 32, 16, 48, 8, 40, 24, 56, 4, 36, 20, 52, 12, 44, 28,60,2, 
 34, 18, 50, 10, 42, 26, 58, 6, 38, 22, 54, 14, 46, 30, 62, 1, 33, 17, 49, 9,41,
 25, 57, 5, 37, 21, 53, 13, 45, 29, 61, 3, 35, 19, 51, 11, 43, 27, 59, 7, 39, 
 23, 55, 15, 47, 31, 63};

__device__ __constant__ int  p128[128] =
{0, 64, 32, 96, 16, 80, 48, 112, 8, 72, 40, 104, 24, 
 88, 56, 120, 4, 68, 36, 100, 20, 84, 52, 116, 12, 76, 44, 108, 28, 92, 60, 
 124, 2, 66, 34, 98, 18, 82, 50, 114, 10, 74, 42, 106, 26, 90, 58, 122, 6, 
 70, 38, 102, 22, 86, 54, 118, 14, 78, 46, 110, 30, 94, 62, 126, 1, 65, 33, 
 97, 17, 81, 49, 113, 9, 73, 41, 105, 25, 89, 57, 121, 5, 69, 37, 101, 21, 
 85, 53, 117, 13, 77, 45, 109, 29, 93, 61, 125, 3, 67, 35, 99, 19, 83, 51, 
 115, 11, 75, 43, 107, 27, 91, 59, 123, 7, 71, 39, 103, 23, 87, 55, 119, 15, 
 79, 47, 111, 31, 95, 63, 127};

__device__ int  p256[256] =
{0, 128, 64, 192, 32, 160, 96, 224, 16, 144, 80, 208, 
 48, 176, 112, 240, 8, 136, 72, 200, 40, 168, 104, 232, 24, 152, 88, 216, 56, 
 184, 120, 248, 4, 132, 68, 196, 36, 164, 100, 228, 20, 148, 84, 212, 52, 180, 
 116, 244, 12, 140, 76, 204, 44, 172, 108, 236, 28, 156, 92, 220, 60, 188, 124,
 252, 2, 130, 66, 194, 34, 162, 98, 226, 18, 146, 82, 210, 50, 178, 114, 242, 
 10, 138, 74, 202, 42, 170, 106, 234, 26, 154, 90, 218, 58, 186, 122, 250, 6, 
 134, 70, 198, 38, 166, 102, 230, 22, 150, 86, 214, 54, 182, 118, 246, 14, 142,
 78, 206, 46, 174, 110, 238, 30, 158, 94, 222, 62, 190, 126, 254, 1, 129, 65, 
 193, 33, 161, 97, 225, 17, 145, 81, 209, 49, 177, 113, 241, 9, 137, 73, 201, 
 41, 169, 105, 233, 25, 153, 89, 217, 57, 185, 121, 249, 5, 133, 69, 197, 37, 
 165, 101, 229, 21, 149, 85, 213, 53, 181, 117, 245, 13, 141, 77, 205, 45, 173,
 109, 237, 29, 157, 93, 221, 61, 189, 125, 253, 3, 131, 67, 195, 35, 163, 99, 
 227, 19, 147, 83, 211, 51, 179, 115, 243, 11, 139, 75, 203, 43, 171, 107, 235,
 27, 155, 91, 219, 59, 187, 123, 251, 7, 135, 71, 199, 39, 167, 103, 231, 23, 
 151, 87, 215, 55, 183, 119, 247, 15, 143, 79, 207, 47, 175, 111, 239, 31, 159,
 95, 223, 63, 191, 127, 255};


/************************************************************************/
__device__ int *ptable(int n)
{
  switch (n)
    {
    case 2: return p2;
    case 4: return p4;
    case 8: return p8;
    case 16: return p16;
    case 32: return p32;
    case 64: return p64;
    case 128: return p128;
    case 256: return p256;
    default: return 0;
    }
}
/************************************************************************/
/* performs a bit-reversal on the data, a double array of size n */
/* Needs workspace of size n */

__device__ void bitreverse(double *data,
		int n,
		double *workspace)
{
  
  int i;
  int *perm;

  perm = ptable(n);
  
  for (i=0; i<n; i++)
    workspace[i] = data[perm[i]];

  memcpy(data, workspace, sizeof(double) * n);

}

